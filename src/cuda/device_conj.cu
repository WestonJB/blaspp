#include "hip/hip_runtime.h"
#include "blas/device.hh"
#include "thrust/complex.h"

#if defined(BLAS_HAVE_CUBLAS)

namespace blas {

template <typename scalar_t>
__global__ void conj_kernel(
    int64_t n,
    scalar_t* src, int64_t inc_src,
    scalar_t* dst, int64_t inc_dst)
{
    using thrust::conj;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        dst[ i*inc_dst ] = conj( src[ i*inc_src ] );
}

//------------------------------------------------------------------------------
/// Conjugates each element of the vector src and stores in dst.
///
/// @param[in] n
///     Number of elements in the vector. n >= 0.
///
/// @param[in] src
///     Pointer to the input vector of length n.
///
/// @param[in] inc_src
///     Stride between elements of src. inc_src >= 1.
///
/// @param[out] dst
///     Pointer to output vector
///     On exit, each element dst[i] is updated as dst[i] = conj( src[i] ).
//      dst may be the same as src.
///
/// @param[in] inc_dst
///     Stride between elements of dst. inc_dst >= 1.
///
/// @param[in] queue
///     BLAS++ queue to execute in.
///
template <typename scalar_t>
void conj(
    int64_t n,
    scalar_t* src, int64_t inc_src,
    scalar_t* dst, int64_t inc_dst,
    blas::Queue& queue )
{
    if (n <= 0) {
        return;
    }

    const int BlockSize = 128;

    int64_t n_threads = std::min( int64_t( BlockSize ), n );
    int64_t n_blocks = 1 + ((n - 1) / n_threads);

    blas_dev_call(
        hipSetDevice( queue.device() ) );

    conj_kernel<<<n_blocks, n_threads, 0, queue.stream()>>>(
        n, src, inc_src, dst, inc_dst );

    blas_dev_call(
        hipGetLastError() );
}

} // namespace blas

#endif // BLAS_HAVE_CUBLAS
